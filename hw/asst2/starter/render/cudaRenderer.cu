#include "hip/hip_runtime.h"
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/sort.h>
#include <thrust/device_vector.h>  // use device_ptr

#include "cudaRenderer.h"
#include "image.h"
#include "noise.h"
#include "sceneLoader.h"
#include "util.h"
#include "cycleTimer.h"

#include "circleBoxTest.cu_inl"

//#define DEBUG

////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

// some contants
#define BOX_SIZE 16

struct GlobalConstants {

    SceneName sceneName;

    int numCircles;
    float* position;
    float* velocity;
    float* color;
    float* radius;

    int imageWidth;
    int imageHeight;
    float* imageData;
};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;

// read-only lookup tables used to quickly compute noise (needed by
// advanceAnimation for the snowflake scene)
__constant__ int    cuConstNoiseYPermutationTable[256];
__constant__ int    cuConstNoiseXPermutationTable[256];
__constant__ float  cuConstNoise1DValueTable[256];

// color ramp table needed for the color ramp lookup shader
#define COLOR_MAP_SIZE 5
__constant__ float  cuConstColorRamp[COLOR_MAP_SIZE][3];


// including parts of the CUDA code from external files to keep this
// file simpler and to seperate code that should not be modified
#include "noiseCuda.cu_inl"
#include "lookupColor.cu_inl"


// kernelClearImageSnowflake -- (CUDA device code)
//
// Clear the image, setting the image to the white-gray gradation that
// is used in the snowflake image
__global__ void kernelClearImageSnowflake() {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float shade = .4f + .45f * static_cast<float>(height-imageY) / height;
    float4 value = make_float4(shade, shade, shade, 1.f);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelClearImage --  (CUDA device code)
//
// Clear the image, setting all pixels to the specified color rgba
__global__ void kernelClearImage(float r, float g, float b, float a) {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelAdvanceFireWorks
// 
// Update the position of the fireworks (if circle is firework)
__global__ void kernelAdvanceFireWorks() {
    const float dt = 1.f / 60.f;
    const float pi = 3.14159;
    const float maxDist = 0.25f;

    float* velocity = cuConstRendererParams.velocity;
    float* position = cuConstRendererParams.position;
    float* radius = cuConstRendererParams.radius;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numCircles)
        return;

    if (0 <= index && index < NUM_FIREWORKS) { // firework center; no update 
        return;
    }

    // determine the fire-work center/spark indices
    int fIdx = (index - NUM_FIREWORKS) / NUM_SPARKS;
    int sfIdx = (index - NUM_FIREWORKS) % NUM_SPARKS;

    int index3i = 3 * fIdx;
    int sIdx = NUM_FIREWORKS + fIdx * NUM_SPARKS + sfIdx;
    int index3j = 3 * sIdx;

    float cx = position[index3i];
    float cy = position[index3i+1];

    // update position
    position[index3j] += velocity[index3j] * dt;
    position[index3j+1] += velocity[index3j+1] * dt;

    // fire-work sparks
    float sx = position[index3j];
    float sy = position[index3j+1];

    // compute vector from firework-spark
    float cxsx = sx - cx;
    float cysy = sy - cy;

    // compute distance from fire-work 
    float dist = sqrt(cxsx * cxsx + cysy * cysy);
    if (dist > maxDist) { // restore to starting position 
        // random starting position on fire-work's rim
        float angle = (sfIdx * 2 * pi)/NUM_SPARKS;
        float sinA = sin(angle);
        float cosA = cos(angle);
        float x = cosA * radius[fIdx];
        float y = sinA * radius[fIdx];

        position[index3j] = position[index3i] + x;
        position[index3j+1] = position[index3i+1] + y;
        position[index3j+2] = 0.0f;

        // travel scaled unit length 
        velocity[index3j] = cosA/5.0;
        velocity[index3j+1] = sinA/5.0;
        velocity[index3j+2] = 0.0f;
    }
}

// kernelAdvanceHypnosis   
//
// Update the radius/color of the circles
__global__ void kernelAdvanceHypnosis() { 
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numCircles) 
        return; 

    float* radius = cuConstRendererParams.radius; 

    float cutOff = 0.5f;
    // place circle back in center after reaching threshold radisus 
    if (radius[index] > cutOff) { 
        radius[index] = 0.02f; 
    } else { 
        radius[index] += 0.01f; 
    }   
}   


// kernelAdvanceBouncingBalls
// 
// Update the positino of the balls
__global__ void kernelAdvanceBouncingBalls() { 
    const float dt = 1.f / 60.f;
    const float kGravity = -2.8f; // sorry Newton
    const float kDragCoeff = -0.8f;
    const float epsilon = 0.001f;

    int index = blockIdx.x * blockDim.x + threadIdx.x; 
   
    if (index >= cuConstRendererParams.numCircles) 
        return; 

    float* velocity = cuConstRendererParams.velocity; 
    float* position = cuConstRendererParams.position; 

    int index3 = 3 * index;
    // reverse velocity if center position < 0
    float oldVelocity = velocity[index3+1];
    float oldPosition = position[index3+1];

    if (oldVelocity == 0.f && oldPosition == 0.f) { // stop-condition 
        return;
    }

    if (position[index3+1] < 0 && oldVelocity < 0.f) { // bounce ball 
        velocity[index3+1] *= kDragCoeff;
    }

    // update velocity: v = u + at (only along y-axis)
    velocity[index3+1] += kGravity * dt;

    // update positions (only along y-axis)
    position[index3+1] += velocity[index3+1] * dt;

    if (fabsf(velocity[index3+1] - oldVelocity) < epsilon
        && oldPosition < 0.0f
        && fabsf(position[index3+1]-oldPosition) < epsilon) { // stop ball 
        velocity[index3+1] = 0.f;
        position[index3+1] = 0.f;
    }
}

// kernelAdvanceSnowflake -- (CUDA device code)
//
// move the snowflake animation forward one time step.  Updates circle
// positions and velocities.  Note how the position of the snowflake
// is reset if it moves off the left, right, or bottom of the screen.
__global__ void kernelAdvanceSnowflake() {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numCircles)
        return;

    const float dt = 1.f / 60.f;
    const float kGravity = -1.8f; // sorry Newton
    const float kDragCoeff = 2.f;

    int index3 = 3 * index;

    float* positionPtr = &cuConstRendererParams.position[index3];
    float* velocityPtr = &cuConstRendererParams.velocity[index3];

    // loads from global memory
    float3 position = *((float3*)positionPtr);
    float3 velocity = *((float3*)velocityPtr);

    // hack to make farther circles move more slowly, giving the
    // illusion of parallax
    float forceScaling = fmin(fmax(1.f - position.z, .1f), 1.f); // clamp

    // add some noise to the motion to make the snow flutter
    float3 noiseInput;
    noiseInput.x = 10.f * position.x;
    noiseInput.y = 10.f * position.y;
    noiseInput.z = 255.f * position.z;
    float2 noiseForce = cudaVec2CellNoise(noiseInput, index);
    noiseForce.x *= 7.5f;
    noiseForce.y *= 5.f;

    // drag
    float2 dragForce;
    dragForce.x = -1.f * kDragCoeff * velocity.x;
    dragForce.y = -1.f * kDragCoeff * velocity.y;

    // update positions
    position.x += velocity.x * dt;
    position.y += velocity.y * dt;

    // update velocities
    velocity.x += forceScaling * (noiseForce.x + dragForce.y) * dt;
    velocity.y += forceScaling * (kGravity + noiseForce.y + dragForce.y) * dt;

    float radius = cuConstRendererParams.radius[index];

    // if the snowflake has moved off the left, right or bottom of
    // the screen, place it back at the top and give it a
    // pseudorandom x position and velocity.
    if ( (position.y + radius < 0.f) ||
         (position.x + radius) < -0.f ||
         (position.x - radius) > 1.f)
    {
        noiseInput.x = 255.f * position.x;
        noiseInput.y = 255.f * position.y;
        noiseInput.z = 255.f * position.z;
        noiseForce = cudaVec2CellNoise(noiseInput, index);

        position.x = .5f + .5f * noiseForce.x;
        position.y = 1.35f + radius;

        // restart from 0 vertical velocity.  Choose a
        // pseudo-random horizontal velocity.
        velocity.x = 2.f * noiseForce.y;
        velocity.y = 0.f;
    }

    // store updated positions and velocities to global memory
    *((float3*)positionPtr) = position;
    *((float3*)velocityPtr) = velocity;
}

// shadePixel -- (CUDA device code)
//
// given a pixel and a circle, determines the contribution to the
// pixel from the circle.  Update of the image is done in this
// function.  Called by kernelRenderCircles()
__device__ __inline__ void
shadePixel(int circleIndex, float2 pixelCenter, float3 p, float4* imagePtr) {

    float diffX = p.x - pixelCenter.x;
    float diffY = p.y - pixelCenter.y;
    float pixelDist = diffX * diffX + diffY * diffY;

    float rad = cuConstRendererParams.radius[circleIndex];;
    float maxDist = rad * rad;

    // circle does not contribute to the image
    if (pixelDist > maxDist)
        return;

    float3 rgb;
    float alpha;

    // there is a non-zero contribution.  Now compute the shading value

    // suggestion: This conditional is in the inner loop.  Although it
    // will evaluate the same for all threads, there is overhead in
    // setting up the lane masks etc to implement the conditional.  It
    // would be wise to perform this logic outside of the loop next in
    // kernelRenderCircles.  (If feeling good about yourself, you
    // could use some specialized template magic).
    if (cuConstRendererParams.sceneName == SNOWFLAKES || cuConstRendererParams.sceneName == SNOWFLAKES_SINGLE_FRAME) {

        const float kCircleMaxAlpha = .5f;
        const float falloffScale = 4.f;

        float normPixelDist = sqrt(pixelDist) / rad;
        rgb = lookupColor(normPixelDist);

        float maxAlpha = .6f + .4f * (1.f-p.z);
        maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.f), 0.f); // kCircleMaxAlpha * clamped value
        alpha = maxAlpha * exp(-1.f * falloffScale * normPixelDist * normPixelDist);

    } else {
        // simple: each circle has an assigned color
        int index3 = 3 * circleIndex;
        rgb = *(float3*)&(cuConstRendererParams.color[index3]);
        alpha = .5f;
    }

    float oneMinusAlpha = 1.f - alpha;

    // BEGIN SHOULD-BE-ATOMIC REGION
    // global memory read

    float4 existingColor = *imagePtr;
    float4 newColor;
    newColor.x = alpha * rgb.x + oneMinusAlpha * existingColor.x;
    newColor.y = alpha * rgb.y + oneMinusAlpha * existingColor.y;
    newColor.z = alpha * rgb.z + oneMinusAlpha * existingColor.z;
    newColor.w = alpha + existingColor.w;

    // global memory write
    *imagePtr = newColor;

    // END SHOULD-BE-ATOMIC REGION
}

// kernelRenderCircles -- (CUDA device code)
//
// Each thread renders a circle.  Since there is no protection to
// ensure order of update or mutual exclusion on the output image, the
// resulting image will be incorrect.
__global__ void kernelRenderCircles(int*& pixelIndex, int*& circleIndex) { 
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numCircles)
        return;

    int index3 = 3 * index;

    // read position and radius
    float3 p = *(float3*)(&cuConstRendererParams.position[index3]);
    float  rad = cuConstRendererParams.radius[index];

    // compute the bounding box of the circle. The bound is in integer
    // screen coordinates, so it's clamped to the edges of the screen.
    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight;
    short minX = static_cast<short>(imageWidth * (p.x - rad));
    short maxX = static_cast<short>(imageWidth * (p.x + rad)) + 1;
    short minY = static_cast<short>(imageHeight * (p.y - rad));
    short maxY = static_cast<short>(imageHeight * (p.y + rad)) + 1;

    // a bunch of clamps.  Is there a CUDA built-in for this?
    short screenMinX = (minX > 0) ? ((minX < imageWidth) ? minX : imageWidth) : 0;
    short screenMaxX = (maxX > 0) ? ((maxX < imageWidth) ? maxX : imageWidth) : 0;
    short screenMinY = (minY > 0) ? ((minY < imageHeight) ? minY : imageHeight) : 0;
    short screenMaxY = (maxY > 0) ? ((maxY < imageHeight) ? maxY : imageHeight) : 0;

    float invWidth = 1.f / imageWidth;
    float invHeight = 1.f / imageHeight;

    // for all pixels in the bonding box
    for (int pixelY=screenMinY; pixelY<screenMaxY; pixelY++) {
        float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * (pixelY * imageWidth + screenMinX)]);
        for (int pixelX=screenMinX; pixelX<screenMaxX; pixelX++) {
            float2 pixelCenterNorm = make_float2(invWidth * (static_cast<float>(pixelX) + 0.5f),
                                                 invHeight * (static_cast<float>(pixelY) + 0.5f));
            shadePixel(index, pixelCenterNorm, p, imgPtr);
            imgPtr++;
        }
    }
}

/************** my bin solution **************/
// myShadePixel -- (CUDA device code)
//
// given a pixel and a circle, determines the contribution to the
// pixel from the circle.  Update of the image is done in this
// function.  Called by kernelRenderCircles()
__device__ __inline__ void
myShadePixel(int circleIndex, float2 pixelCenter, float3 p, float4* imagePtr) {

    float diffX = p.x - pixelCenter.x;
    float diffY = p.y - pixelCenter.y;
    float pixelDist = diffX * diffX + diffY * diffY;

    float rad = cuConstRendererParams.radius[circleIndex];;

    float3 rgb;
    float alpha;

    // there is a non-zero contribution.  Now compute the shading value

    // suggestion: This conditional is in the inner loop.  Although it
    // will evaluate the same for all threads, there is overhead in
    // setting up the lane masks etc to implement the conditional.  It
    // would be wise to perform this logic outside of the loop next in
    // kernelRenderCircles.  (If feeling good about yourself, you
    // could use some specialized template magic).
    if (cuConstRendererParams.sceneName == SNOWFLAKES || cuConstRendererParams.sceneName == SNOWFLAKES_SINGLE_FRAME) {

        const float kCircleMaxAlpha = .5f;
        const float falloffScale = 4.f;

        float normPixelDist = sqrt(pixelDist) / rad;
        rgb = lookupColor(normPixelDist);

        float maxAlpha = .6f + .4f * (1.f-p.z);
        maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.f), 0.f); // kCircleMaxAlpha * clamped value
        alpha = maxAlpha * exp(-1.f * falloffScale * normPixelDist * normPixelDist);

    } else {
        // simple: each circle has an assigned color
        int index3 = 3 * circleIndex;
        rgb = *(float3*)&(cuConstRendererParams.color[index3]);
        alpha = .5f;
    }

    float oneMinusAlpha = 1.f - alpha;

    // BEGIN SHOULD-BE-ATOMIC REGION
    // global memory read

    float4 existingColor = *imagePtr;
    float4 newColor;
    newColor.x = alpha * rgb.x + oneMinusAlpha * existingColor.x;
    newColor.y = alpha * rgb.y + oneMinusAlpha * existingColor.y;
    newColor.z = alpha * rgb.z + oneMinusAlpha * existingColor.z;
    newColor.w = alpha + existingColor.w;

    // global memory write
    *imagePtr = newColor;

    // END SHOULD-BE-ATOMIC REGION
}

#ifdef SOLUTION2
/************* solution 2: render by circles and sort ****************/
__global__ void kernelInitValue(int* binPixelIndex, int initVal, int size) { 
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size) {
        return;
    }
    binPixelIndex[index] = initVal;
}

__global__ void myBinKernelRenderCircles(int* binCircleIndex, int* begin, int* end, int pixelNum) {
    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height) {
        return;
    }

    int invWidth = 1.f / width;
    int invHeight = 1.f / height;
    
    float2 pixelCenterNorm = make_float2(
        invWidth * (static_cast<float>(imageX) + 0.5f),
        invHeight * (static_cast<float>(imageY) + 0.5f));
    int pixelIndex = imageY * width + imageX;
    int offset = 4 * pixelIndex;
    float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[offset]);

    int b = begin[pixelIndex];
    int e = end[pixelIndex];

    if (b == e) {
        return;
    }

    //printf("%d\t%d\n", b, e);

    for (int i = b; i < e; ++i) {
        int index = binCircleIndex[i];
//        printf("%d\n", index);
        int index3 = 3 * index;
        float3 p = *(float3*)(&cuConstRendererParams.position[index3]);
        myShadePixel(index, pixelCenterNorm, p, imgPtr);
    }
}

__global__ void kernelFindBeginAndEnd(int* binPixelIndex, int* begin, int* end, int totalSize, int pixelNum) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= totalSize || binPixelIndex[index] == pixelNum) {
        return;
    }
    int pixelIndex = binPixelIndex[index];
    if (index == 0 || pixelIndex != binPixelIndex[index - 1]) {
        begin[pixelIndex] = index;
#ifdef DEBUG
        if (index % 768 == 0) {
            printf("%d\t%d\n", index, pixelIndex);
        }
#endif
    }
    if (index == totalSize - 1 || pixelIndex != binPixelIndex[index + 1]) { 
        end[pixelIndex] = index + 1;
    }
}

__global__ void kernelFindBin(int* binPixelIndex, int* binCircleIndex) { 
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numCircles)
        return;

    int index3 = 3 * index;

    // read position and radius
    float3 p = *(float3*)(&cuConstRendererParams.position[index3]);
    float  rad = cuConstRendererParams.radius[index];

    // compute the bounding box of the circle. The bound is in integer
    // screen coordinates, so it's clamped to the edges of the screen.
    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight;
    short minX = static_cast<short>(imageWidth * (p.x - rad));
    short maxX = static_cast<short>(imageWidth * (p.x + rad)) + 1;
    short minY = static_cast<short>(imageHeight * (p.y - rad));
    short maxY = static_cast<short>(imageHeight * (p.y + rad)) + 1;

    // a bunch of clamps.  Is there a CUDA built-in for this?
    short screenMinX = (minX > 0) ? ((minX < imageWidth) ? minX : imageWidth) : 0;
    short screenMaxX = (maxX > 0) ? ((maxX < imageWidth) ? maxX : imageWidth) : 0;
    short screenMinY = (minY > 0) ? ((minY < imageHeight) ? minY : imageHeight) : 0;
    short screenMaxY = (maxY > 0) ? ((maxY < imageHeight) ? maxY : imageHeight) : 0;

    float invWidth = 1.f / imageWidth;
    float invHeight = 1.f / imageHeight;

    int binIndex = index * imageWidth * imageHeight;

    // for all pixels in the bonding box
    for (int pixelY=screenMinY; pixelY<screenMaxY; pixelY++) {
        float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * (pixelY * imageWidth + screenMinX)]);
        for (int pixelX=screenMinX; pixelX<screenMaxX; pixelX++) {
            float2 pixelCenterNorm = make_float2(invWidth * (static_cast<float>(pixelX) + 0.5f),
                                                 invHeight * (static_cast<float>(pixelY) + 0.5f));
            float diffX = p.x - pixelCenterNorm.x;
            float diffY = p.y - pixelCenterNorm.y;
            float pixelDist = diffX * diffX + diffY * diffY;
            float rad = cuConstRendererParams.radius[index];
            float maxDist = rad * rad;
            if (pixelDist <= maxDist) {
                binCircleIndex[binIndex] = index;
                int pixelIndex = pixelY * imageWidth + pixelX;
                binPixelIndex[binIndex++] = pixelIndex;
            }
            imgPtr++;
        }
    }
}
/********************* end of solution 2 ***************************/
#endif

#ifdef SOLUTION1
/************** Soluton 1: render by cells ********************/
// myKernelRenderCells-- (CUDA device code)
//
// Each thread renders a cell. Loop through all circles to decide 
// if it makes a contribution to the cell. 
// Note: This method may be inefficient when the number of cells
//       far less than the number of circles.  
__global__ void myKernelRenderCells() {
    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    // check boundary
    if (imageX >= width || imageY >= height) {
        return;
    }

    float invWidth = 1.f / width;
    float invHeight = 1.f / height;

    float2 pixelCenterNorm = make_float2(
        invWidth * (static_cast<float>(imageX) + 0.5f),
        invHeight * (static_cast<float>(imageY) + 0.5f));
    int offset = 4 * (imageY * width + imageX);
    float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[offset]);
    
    // loop through all circles
    int circleNum = cuConstRendererParams.numCircles;
    for (int circleIndex = 0; circleIndex < circleNum; ++circleIndex) {
        int index3 = circleIndex * 3;
        float3 p = *(float3*)(&cuConstRendererParams.position[index3]);
        shadePixel(circleIndex, pixelCenterNorm, p, imgPtr);
    }
}
/**************** End of code solution 1: render by cells ******************/
#endif

#ifdef SOLUTION3
/************** solution 3: render by blocks(bloom filter) ****************/
__global__ void getCirclesInBox(int* cudaDeviceCirclesInBox, int* cudaDeviceBoxCirclesCount, 
    int boxNum_x, int boxNum_y) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= boxNum_x || y >= boxNum_y) {
        return;
    }

    short width = cuConstRendererParams.imageWidth;
    short height = cuConstRendererParams.imageHeight;

    float invWidth = 1.f / width;
    float invHeight = 1.f / height;


    // get the boundary of the box
    float boxL = invWidth * static_cast<float>(x * BOX_SIZE);
    float boxR = invWidth * static_cast<float>((x + 1) * BOX_SIZE);
    float boxT = invHeight * static_cast<float>((y + 1) * BOX_SIZE);
    float boxB = invHeight * static_cast<float>(y * BOX_SIZE);

    int circlesNum = cuConstRendererParams.numCircles;
    int boxIndex = boxNum_x * y + x;
    int boxOffset = circlesNum * boxIndex;
    int circlesCount = 0;
    // walk through all circles to find those intersect with the box
    for (int index = 0; index < circlesNum; ++index) {
        int index3 = index * 3;
        float3 p = *(float3*)(&cuConstRendererParams.position[index3]);
        float rad = cuConstRendererParams.radius[index];

        if (circleInBox(p.x, p.y, rad, boxL, boxR, boxT, boxB)) {
            cudaDeviceCirclesInBox[boxOffset + circlesCount] = index;
            ++circlesCount;
        }
    }

    //printf("circle count %d\n", circlesCount);
    cudaDeviceBoxCirclesCount[boxIndex] = circlesCount;
}

__global__ void kernelRenderCirclesByBox(int* cudaDeviceCirclesInBox, int* cudaDeviceBoxCirclesCount, 
    int boxNum_x, int boxNum_y) {
    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    short width = cuConstRendererParams.imageWidth;
    short height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height) {
        return;
    }

    float invWidth = 1.f / width;
    float invHeight = 1.f / height;

    float2 pixelCenterNorm = make_float2(
        invWidth * (static_cast<float>(imageX) + 0.5f),
        invHeight * (static_cast<float>(imageY) + 0.5f));
    int offset = 4 * (imageY * width + imageX);
    float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[offset]);
    
    // loop through all circles in the box
    int index_x = imageX / BOX_SIZE; 
    int index_y = imageY / BOX_SIZE;
    int boxIndex = index_y * boxNum_x + index_x;
    int circlesInBoxNum = cudaDeviceBoxCirclesCount[boxIndex];
    int numCircles = cuConstRendererParams.numCircles;
    for (int i = 0; i < circlesInBoxNum; ++i) {
        int index = cudaDeviceCirclesInBox[numCircles * boxIndex + i]; 
        int index3 = index * 3;
        float3 p = *(float3*)(&cuConstRendererParams.position[index3]);
        shadePixel(index, pixelCenterNorm, p, imgPtr);
    }
}
/********************* End of solution 3: render by box ******************/
#endif

////////////////////////////////////////////////////////////////////////////////////////

CudaRenderer::CudaRenderer() {
    image = NULL;

    numCircles = 0;
    position = NULL;
    velocity = NULL;
    color = NULL;
    radius = NULL;

#ifdef SOLUTION3
    cudaDeviceCirclesInBox = NULL;
    cudaDeviceBoxCirclesCount = NULL;
#endif

#ifdef SOLUTION2
    binPixelIndex = NULL;
    binCircleIndex = NULL;
    begin = NULL;
    end = NULL;
#endif
    cudaDevicePosition = NULL;
    cudaDeviceVelocity = NULL;
    cudaDeviceColor = NULL;
    cudaDeviceRadius = NULL;
    cudaDeviceImageData = NULL;
}

CudaRenderer::~CudaRenderer() {

    if (image) {
        delete image->data;   // delete data first!!
        delete image;
    }

    if (position) {
        delete [] position;
        delete [] velocity;
        delete [] color;
        delete [] radius;
    }

    if (cudaDevicePosition) {
        printf("free space!\n");
#ifdef SOLUTION3
        hipFree(cudaDeviceCirclesInBox);
        hipFree(cudaDeviceBoxCirclesCount);
#endif

#ifdef SOLUTION2
        hipFree(binPixelIndex);
        hipFree(binCircleIndex);
        hipFree(begin);
        hipFree(end);
#endif
        hipFree(cudaDevicePosition);
        hipFree(cudaDeviceVelocity);
        hipFree(cudaDeviceColor);
        hipFree(cudaDeviceRadius);
        hipFree(cudaDeviceImageData);
    }
    hipFree(cuConstNoiseYPermutationTable);
    hipFree(cuConstNoiseXPermutationTable);
    hipFree(cuConstNoise1DValueTable);
}

const Image*
CudaRenderer::getImage() {

    // need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    printf("Copying image data from device\n");

    hipMemcpy(image->data,
               cudaDeviceImageData,
               sizeof(float) * 4 * image->width * image->height,
               hipMemcpyDeviceToHost);

    return image;
}

void
CudaRenderer::loadScene(SceneName scene) {
    sceneName = scene;
    loadCircleScene(sceneName, numCircles, position, velocity, color, radius);
}

void
CudaRenderer::setup() {

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
    
    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy


#ifdef SOLUTION3
    short boxNum_x = (image->width + BOX_SIZE - 1) / BOX_SIZE;
    short boxNum_y = (image->height + BOX_SIZE - 1) / BOX_SIZE;
    hipMalloc(&cudaDeviceCirclesInBox, sizeof(int) * numCircles * boxNum_x * boxNum_y);
    hipMalloc(&cudaDeviceBoxCirclesCount, sizeof(int) * boxNum_x * boxNum_y);
#endif

#ifdef SOLUTION2
    int totalPixelNum = image->width * image->height;
    int totalSize = numCircles * totalPixelNum;
    hipMalloc(&binPixelIndex, sizeof(int) * totalSize);
    hipMalloc(&binCircleIndex, sizeof(int) * totalSize);
    hipMalloc(&begin, sizeof(int) * totalPixelNum);
    hipMalloc(&end, sizeof(int) * totalPixelNum);
    // initialize to max, so that after sorting it appears at last
    dim3 blockDim1(256, 1);
    dim3 gridDim1((totalSize + blockDim1.x - 1) / blockDim1.x);
    kernelInitValue<<<gridDim1, blockDim1>>>(binPixelIndex, totalPixelNum, totalSize);
    // find the start and end index of each bin
    dim3 blockDim2(256, 1);
    dim3 gridDim2((totalPixelNum + blockDim2.x - 1) / blockDim2.x);
    kernelInitValue<<<gridDim2, blockDim2>>>(begin, -1, totalPixelNum);
    kernelInitValue<<<gridDim2, blockDim2>>>(end, -1, totalPixelNum);
#endif

    hipMalloc(&cudaDevicePosition, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceVelocity, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceRadius, sizeof(float) * numCircles);
    hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height);

    hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numCircles, hipMemcpyHostToDevice);

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;
    params.sceneName = sceneName;
    params.numCircles = numCircles;
    params.imageWidth = image->width;
    params.imageHeight = image->height;
    params.position = cudaDevicePosition;
    params.velocity = cudaDeviceVelocity;
    params.color = cudaDeviceColor;
    params.radius = cudaDeviceRadius;
    params.imageData = cudaDeviceImageData;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

    // also need to copy over the noise lookup tables, so we can
    // implement noise on the GPU
    int* permX;
    int* permY;
    float* value1D;
    getNoiseTables(&permX, &permY, &value1D);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

    // last, copy over the color table that's used by the shading
    // function for circles in the snowflake demo

    float lookupTable[COLOR_MAP_SIZE][3] = {
        {1.f, 1.f, 1.f},
        {1.f, 1.f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, 0.8f, 1.f},
    };

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);

}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void
CudaRenderer::allocOutputImage(int width, int height) {

    if (image) {
        delete image->data;   // delete data as well!!!!
        delete image;
    }
    image = new Image(width, height);
}

// clearImage --
//
// Clear's the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void
CudaRenderer::clearImage() {

    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
        (image->width + blockDim.x - 1) / blockDim.x,
        (image->height + blockDim.y - 1) / blockDim.y);

    if (sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME) {
        kernelClearImageSnowflake<<<gridDim, blockDim>>>();
    } else {
        kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
    }
    hipDeviceSynchronize();
}

// advanceAnimation --
//
// Advance the simulation one time step.  Updates all circle positions
// and velocities
void
CudaRenderer::advanceAnimation() {
     // 256 threads per block is a healthy number
    dim3 blockDim(256, 1);
    dim3 gridDim((numCircles + blockDim.x - 1) / blockDim.x);

    // only the snowflake scene has animation
    if (sceneName == SNOWFLAKES) {
        kernelAdvanceSnowflake<<<gridDim, blockDim>>>();
    } else if (sceneName == BOUNCING_BALLS) {
        kernelAdvanceBouncingBalls<<<gridDim, blockDim>>>();
    } else if (sceneName == HYPNOSIS) {
        kernelAdvanceHypnosis<<<gridDim, blockDim>>>();
    } else if (sceneName == FIREWORKS) { 
        kernelAdvanceFireWorks<<<gridDim, blockDim>>>(); 
    }
    hipDeviceSynchronize();
}

void
CudaRenderer::render() {
#ifdef SOLUTION1
    // solution 1: create threads acoording to cells, get 21/65
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
        (image->width + blockDim.x - 1) / blockDim.x, 
        (image->height + blockDim.y - 1) / blockDim.y);

    myKernelRenderCells<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
#endif

#ifdef SOLUTION2
    // solution 2: data-parallel approach, get 2/65, out of memory
    int totalPixelNum = image->width * image->height;
    int totalSize = numCircles * totalPixelNum;

    // put all pixel into circle bins
    double startTime = CycleTimer::currentSeconds();
    dim3 blockDim1(256, 1);
    dim3 gridDim1((numCircles + blockDim1.x - 1) / blockDim1.x);
    kernelFindBin<<<gridDim1, blockDim1>>>(binPixelIndex, binCircleIndex);   
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
    printf("time to put into bin: %.4f ms\n", 1000 * (endTime - startTime));

    // sort bins by pixel index
    // must wrap device memory with device_ptr
    startTime = CycleTimer::currentSeconds();
    thrust::device_ptr<int> dev_binPixelIndex(binPixelIndex);
    thrust::device_ptr<int> dev_binCircleIndex(binCircleIndex);
    thrust::stable_sort_by_key(dev_binPixelIndex, dev_binPixelIndex + totalSize, dev_binCircleIndex);
    endTime = CycleTimer::currentSeconds();
    printf("time to sort: %.4f ms\n", 1000 * (endTime - startTime));

    startTime = CycleTimer::currentSeconds();
    dim3 blockDim2(256, 1);
    dim3 gridDim2((totalSize + blockDim2.x - 1) / blockDim2.x);
    kernelFindBeginAndEnd<<<gridDim2, blockDim2>>>(binPixelIndex, 
        begin, end, totalSize, totalPixelNum);
    hipDeviceSynchronize();
    endTime = CycleTimer::currentSeconds();
    printf("time to find begin and end: %.4f ms\n", 1000 * (endTime - startTime));

    // render every pixel
    startTime = CycleTimer::currentSeconds();
    dim3 blockDim3(16, 16, 1);
    dim3 gridDim3((image->width + blockDim3.x - 1) / blockDim3.x,
                    (image->height + blockDim3.y - 1) / blockDim3.y);
    myBinKernelRenderCircles<<<gridDim3, blockDim3>>>(binCircleIndex, begin, end, totalPixelNum);
    hipDeviceSynchronize();
    endTime = CycleTimer::currentSeconds();
    printf("time to render by pixel: %.4f ms\n", 1000 * (endTime - startTime));
#endif 

    // solution 3: bloom filter, get 65/65
#ifdef SOLUTION3
    short boxNum_x = (image->width + BOX_SIZE - 1) / BOX_SIZE;
    short boxNum_y = (image->height + BOX_SIZE - 1) / BOX_SIZE;
    dim3 blockDim(8, 8);
    dim3 gridDim((boxNum_x + blockDim.x - 1) / blockDim.x, 
                (boxNum_y + blockDim.y - 1) / blockDim.y);

    getCirclesInBox<<<blockDim, gridDim>>>(cudaDeviceCirclesInBox, cudaDeviceBoxCirclesCount, boxNum_x, boxNum_y);
    hipDeviceSynchronize();
    dim3 blockDim2(16, 16, 1);
    dim3 gridDim2(
        (image->width + blockDim2.x - 1) / blockDim2.x, 
        (image->height + blockDim2.y - 1) / blockDim2.y);

    kernelRenderCirclesByBox<<<gridDim2, blockDim2>>>(cudaDeviceCirclesInBox, cudaDeviceBoxCirclesCount, boxNum_x, boxNum_y);
    hipDeviceSynchronize();
    hipError_t cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess) {
        printf("cada failed: %s\n", hipGetErrorString(cudaerr));
    }
#endif
}
